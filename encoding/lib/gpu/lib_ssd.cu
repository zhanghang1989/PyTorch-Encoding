#include "hip/hip_runtime.h"
/******************************************************************************
*
* Copyright (c) 2018-2019, NVIDIA CORPORATION. All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*

 ******************************************************************************/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCNumerics.cuh>
#include <THC/THC.h>

#include <hip/hip_runtime.h>

//#define DEBUG

// calculate the IoU of a single box against another box
__device__
float calc_single_iou(const float4 b1, const float4 b2) {
  // (lt), (rb)
  float l = max(b1.x, b2.x);
  float t = max(b1.y, b2.y);
  float r = min(b1.z, b2.z);
  float b = min(b1.w, b2.w);

  float first = (r - l);
  first = (first < 0) ? 0 : first;
  float second = (b - t);
  second = (second < 0) ? 0 : second;

  float intersection = first * second;

  float area1 = (b1.w - b1.y) * (b1.z - b1.x);
  float area2 = (b2.w - b2.y) * (b2.z - b2.x);

  return intersection / (area1 + area2 - intersection);
}

__global__
// boxes1 : [N x 4]
// boxes2 : [M x 4]
//   ious : [N x M]
void calc_ious_kernel(const int N_img, const float4 *box1, const int *box1_offsets,
                      const int M, const float4 *boxes2, float *ious) {

  // launch N_img blocks
  const int img = blockIdx.x;

  // each block, i will run over the box1_N[i] source and M target boxes
  // generating box1_N[i] x M outputs

  // alias to start of boxes for this image
  const float4 *b1 = &box1[box1_offsets[img]];

  if (threadIdx.x == 0) {
    //printf("offset for img %d : %d\n", img, box1_offsets[img]);
  }

  // number of boxes for this image from offsets
  int N = box1_offsets[img+1] - box1_offsets[img];

  for (int i = 0; i < N; ++i) {
    // if (threadIdx.x == 0) printf("i : %d\n", i);
    const float4 source = b1[i];
    // for each source, loop over targets
    for (int j = threadIdx.x; j < M; j += blockDim.x) {
      const float4 target = boxes2[j];

      float iou = calc_single_iou(source, target);

      // store the calculated IoU in the correct spot
      int out_idx = box1_offsets[img] * M + i * M + j;
      ious[out_idx] = iou;

    }
  }
}

__device__
void reduce_val_idx(int N, volatile float *vals, volatile int *idx) {
  // naive: single thread for now
  if (threadIdx.x == 0) {
    float max_val = vals[0];
    int max_idx = idx[0];

    for (int i = 1; i < N; ++i) {
      if (vals[i] > max_val) {
        max_val = vals[i];
        max_idx = idx[i];
      }
    }

    vals[0] = max_val;
    idx[0] = max_idx;
  }
}

/**
 * perform remaining parts, storing temporary values in global workspace
 * workspace needs N_img * M values, each of 8 bytes (float, int)
 **/
template <int BLOCK_SIZE, int MAX_BBOXES_PER_BLOCK>
__global__
void encode(const int N_img, const float4 *bbox_in, const long *labels_in, const int *offsets,
            const int M, const float4 *dboxes, // const float *ious,
            const float criteria, uint8_t *workspace, float4 *bbox_out, long *label_out) {

  // Each block will take a single image's IoU set
  const int img = blockIdx.x;

  // shared memory for intermediate results
  __shared__ volatile float best_bbox_iou_tmp[BLOCK_SIZE];
  __shared__ volatile int best_bbox_idx_tmp[BLOCK_SIZE];

  // shared memory for final best_bbox_{iou, idx} values
  __shared__ volatile float best_bbox_iou[MAX_BBOXES_PER_BLOCK];
  __shared__ volatile int best_bbox_idx[MAX_BBOXES_PER_BLOCK];

  // index into the global workspace - each image needs (float + int) * M values
  volatile float *best_dbox_iou = (float *)&workspace[img * M * 8];
  volatile int *best_dbox_idx = (int *)&workspace[img * M * 8 + M * 4];

  // number of input bboxes for this image
  const int N_rows = offsets[img+1] - offsets[img];

  // Check for potential crash
  assert(N_rows <= MAX_BBOXES_PER_BLOCK);
#ifdef DEBUG
  if (threadIdx.x == 0)
    printf("N rows: %d %d to %d (%p - %p)\n", N_rows, offsets[img], offsets[img+1], best_dbox_iou, best_dbox_idx);
#endif

  for (int i = threadIdx.x; i < MAX_BBOXES_PER_BLOCK; i += blockDim.x) {
    best_bbox_iou[i] = -FLT_MAX;
    best_bbox_idx[i] = -1;
  }
  __syncthreads();

  // loop serially over the rows of the IoU set that correspond to this image
  int row_num = 0;
  for (int i = offsets[img]; i < offsets[img+1]; ++i) {
    // reset shmem tallies
    best_bbox_iou_tmp[threadIdx.x] = -FLT_MAX;
    best_bbox_idx_tmp[threadIdx.x] = -1;

    // index into the input buffer
    // const float *row = &ious[i * M];
    const float4 input_bbox = bbox_in[i];
#ifdef DEBUG
    if (threadIdx.x == 0)
      printf("%d - %p\n", img, &input_bbox);
#endif

    // loop by threads over the columns
    for (int j = threadIdx.x; j < M; j += blockDim.x) {

      // check and store new max if necessary
      const float4 input_dbox = dboxes[j];
      // float new_val = row[j];
      float new_val = calc_single_iou(input_bbox, input_dbox);

      // handle per-row max in shared memory
      if (new_val > best_bbox_iou_tmp[threadIdx.x]) {
        best_bbox_iou_tmp[threadIdx.x] = new_val;
        best_bbox_idx_tmp[threadIdx.x] = j;
      }

      // handle per-col max in global workspace
      if (new_val > best_dbox_iou[j]) {
        best_dbox_iou[j] = new_val;
        best_dbox_idx[j] = row_num;

#ifdef DEBUG
        assert(best_dbox_idx[j] >= 0);
        assert(best_dbox_idx[j] < N_rows);
#endif
      }
    }

    // Now we have all the values for this row -- reduce
    __syncthreads();

    // reduce - output is in max_{val, idx}_row[0]
    reduce_val_idx(blockDim.x, best_bbox_iou_tmp, best_bbox_idx_tmp);
#ifdef DEBUG
    __syncthreads();
#endif


    // store output for row i
    if (threadIdx.x == 0) {
      best_bbox_iou[row_num] = best_bbox_iou_tmp[0];
      best_bbox_idx[row_num] = best_bbox_idx_tmp[0];

#ifdef DEBUG
      assert(best_bbox_idx[row_num] >= 0);
      assert(best_bbox_idx[row_num] < M);
#endif
    }
    __syncthreads();

    // keep track of _local_ row
    row_num++;
  }

#ifdef DEBUG
  if (threadIdx.x == 0) {
    for (int i = 0; i < N_rows; ++i) {
      printf("%d - row : %d : best bbox_idx: %d\n", img, i, best_bbox_idx[i]);
    }
  }
#endif

#ifdef DEBUG
  // make sure all best_bbox_{iou, val} are seen by everyone
  __syncthreads();
#endif
  // At this point we have the maximum values & indices for both bbox and dbox
  /*
        best_dbox_ious.index_fill_(0, best_bbox_idx, 2.0)

        idx = torch.arange(0, best_bbox_idx.size(0), dtype=torch.int64)
        best_dbox_idx[best_bbox_idx[idx]] = idx
  */
  for (int i = threadIdx.x; i < N_rows; i += blockDim.x) {
    int idx = best_bbox_idx[i];

#ifdef DEBUG
    assert(idx < M);
    assert(idx >= 0);
#endif

    best_dbox_iou[idx] = 2.;
    best_dbox_idx[idx] = i;
#ifdef DEBUG
    printf("%d - set best dbox_idx[%d] to %d\n", img, best_bbox_idx[i], i);
#endif
  }

  /**
        # filter IoU > 0.5
        masks = best_dbox_ious > criteria
        labels_out = torch.zeros(self.nboxes, dtype=torch.long)
        #print(maxloc.shape, labels_in.shape, labels_out.shape)
        labels_out[masks] = labels_in[best_dbox_idx[masks]]
        bboxes_out = self.dboxes.clone()
        bboxes_out[masks, :] = bboxes_in[best_dbox_idx[masks], :]
        # Transform format to xywh format
        x, y, w, h = 0.5*(bboxes_out[:, 0] + bboxes_out[:, 2]), \
                     0.5*(bboxes_out[:, 1] + bboxes_out[:, 3]), \
                     -bboxes_out[:, 0] + bboxes_out[:, 2], \
                     -bboxes_out[:, 1] + bboxes_out[:, 3]
        bboxes_out[:, 0] = x
        bboxes_out[:, 1] = y
        bboxes_out[:, 2] = w
        bboxes_out[:, 3] = h
        return bboxes_out, labels_out
  **/
  __syncthreads();
  for (int i = threadIdx.x; i < M; i += blockDim.x) {
    // offset into output arrays: M values per image
    // int output_idx = offsets[img] * M + i;
    int output_idx = img * M + i;

    // reset output labels to background
    // NOTE: bbox_out is already cloned from dbox outside of this kernel
    label_out[output_idx] = 0;

    // Filter IoU > 0.5
    bool mask = best_dbox_iou[i] > criteria;

    float4 bbox = bbox_out[output_idx];
    // copy some labels and bboxes
    if (mask) {
      // copy label
#ifdef DEBUG
      printf("%d : label: local input idx: %d, value: %d\n", i, best_dbox_idx[i], labels_in[offsets[img] + best_dbox_idx[i]]);
      // printf("%d : label: local input idx: %d, value: %d\n", i, best_dbox_idx[i], labels_in[offsets[img] + i]);
#endif
      label_out[output_idx] = labels_in[offsets[img] + best_dbox_idx[i]];

      // grab original box
      bbox = bbox_in[offsets[img] + best_dbox_idx[i]];
#ifdef DEBUG
      printf("mask %d : %d : %f %f %f %f\n", i, best_dbox_idx[i], bbox.x, bbox.y, bbox.z, bbox.w);
#endif
    }

    // transfer to xywh
    float4 bbox_tmp;
    bbox_tmp.x = 0.5 * (bbox.x + bbox.z);
    bbox_tmp.y = 0.5 * (bbox.y + bbox.w);
    bbox_tmp.z = bbox.z - bbox.x;
    bbox_tmp.w = bbox.w - bbox.y;

    // write out
    bbox_out[output_idx] = bbox_tmp;
  }
}

/**
    def encode(self, bboxes_in, labels_in, criteria = 0.5):

        ious = calc_iou_tensor(bboxes_in, self.dboxes)
        best_dbox_ious, best_dbox_idx = ious.max(dim=0)
        best_bbox_ious, best_bbox_idx = ious.max(dim=1)

        # set best ious 2.0
        best_dbox_ious.index_fill_(0, best_bbox_idx, 2.0)

        idx = torch.arange(0, best_bbox_idx.size(0), dtype=torch.int64)
        best_dbox_idx[best_bbox_idx[idx]] = idx

        # filter IoU > 0.5
        masks = best_dbox_ious > criteria
        labels_out = torch.zeros(self.nboxes, dtype=torch.long)
        #print(maxloc.shape, labels_in.shape, labels_out.shape)
        labels_out[masks] = labels_in[best_dbox_idx[masks]]
        bboxes_out = self.dboxes.clone()
        bboxes_out[masks, :] = bboxes_in[best_dbox_idx[masks], :]
        # Transform format to xywh format
        x, y, w, h = 0.5*(bboxes_out[:, 0] + bboxes_out[:, 2]), \
                     0.5*(bboxes_out[:, 1] + bboxes_out[:, 3]), \
                     -bboxes_out[:, 0] + bboxes_out[:, 2], \
                     -bboxes_out[:, 1] + bboxes_out[:, 3]
        bboxes_out[:, 0] = x
        bboxes_out[:, 1] = y
        bboxes_out[:, 2] = w
        bboxes_out[:, 3] = h
        return bboxes_out, labels_out
**/
std::vector<at::Tensor> box_encoder(const int N_img,
                                    const at::Tensor& bbox_input,
                                    const at::Tensor& bbox_offsets,
                                    const at::Tensor& labels_input,
                                    const at::Tensor& dbox,
                                    float criteria) {
  // Check everything is on the device
  AT_ASSERTM(bbox_input.is_cuda(), "bboxes must be a CUDA tensor");
  AT_ASSERTM(bbox_offsets.is_cuda(), "bbox offsets must be a CUDA tensor");
  AT_ASSERTM(labels_input.is_cuda(), "labels must be a CUDA tensor");
  AT_ASSERTM(dbox.is_cuda(), "dboxes must be a CUDA tensor");

  // Check at least offsets, bboxes and labels are consistent
  // Note: offsets is N+1 vs. N for labels
  AT_ASSERTM(N_img + 1 == bbox_offsets.numel(), "must have N_img+1 offsets");


  auto num_bbox_total = bbox_offsets[bbox_offsets.numel()-1].item<int>();
#ifdef DEBUG
  printf("%d : bboxes: %d\n", (int)bbox_offsets.numel(), num_bbox_total);
#endif
  AT_ASSERTM(num_bbox_total <= 2048, "total num bboxes must be <= 2048");

  AT_ASSERTM(bbox_input.size(0) == labels_input.size(0), "bbox and labels must have same leading dimension");

  const int N = bbox_input.size(0);
  const int M = dbox.size(0);

  auto stream = at::cuda::getCurrentCUDAStream();

  // allocate final outputs (known size)
#ifdef DEBUG
  printf("%d x %d\n", N_img * M, 4);
  // at::Tensor bbox_out = dbox.scalar_type().tensor({N_img * M, 4});
  printf("allocating %lu bytes for output labels\n", N_img*M*sizeof(long));
#endif
  at::Tensor labels_out = at::empty({N_img * M}, labels_input.options());
  THCudaCheck(hipGetLastError());

  // copy default boxes to outputs
#ifdef DEBUG
  printf("allocating %lu bytes for output bboxes\n", N_img*M*4*sizeof(float));
#endif
  at::Tensor bbox_out = dbox.repeat({N_img, 1});
  THCudaCheck(hipGetLastError());

  // need to allocate some workspace
#ifdef DEBUG
  printf("allocating %lu bytes for workspace\n", 8*M*N_img);
#endif
  // at::Tensor workspace = at::CUDA(at::kByte).zeros({8 * M * N_img});
  at::Tensor workspace = at::zeros({8 * M * N_img}, at::CUDA(at::kByte));
  THCudaCheck(hipGetLastError());

  // Encode the inputs
  const int THREADS_PER_BLOCK = 256;
  encode<THREADS_PER_BLOCK, 256><<<N_img, THREADS_PER_BLOCK, 0, stream.stream()>>>(N_img,
                      (float4*)bbox_input.data_ptr<float>(),
                      labels_input.data_ptr<long>(),
                      bbox_offsets.data_ptr<int>(),
                      M,
                      (float4*)dbox.data_ptr<float>(),
                      criteria,
                      workspace.data_ptr<uint8_t>(),
                      (float4*)bbox_out.data_ptr<float>(),
                      labels_out.data_ptr<long>());

  THCudaCheck(hipGetLastError());
  return {bbox_out, labels_out};
}

at::Tensor calc_ious(const int N_img,
                     const at::Tensor& boxes1,
                     const at::Tensor& boxes1_offsets,
                     const at::Tensor& boxes2) {

  const int N = boxes1.size(0);
  const int M = boxes2.size(0);

  auto stream = at::cuda::getCurrentCUDAStream();

  // at::Tensor ious = at::CUDA(at::kFloat).zeros({N, M});
  // at::Tensor ious = at::ones(at::CUDA(at::kFloat), {N, M});
  at::Tensor ious = at::empty({N, M}, boxes1.options());

  // Get IoU of all source x default box pairs
  calc_ious_kernel<<<N_img, 256, 0, stream.stream()>>>(
                        N_img,
                        (float4*)boxes1.data_ptr<float>(),
                        boxes1_offsets.data_ptr<int>(),
                        M,
                        (float4*)boxes2.data_ptr<float>(),
                        ious.data_ptr<float>());

  THCudaCheck(hipGetLastError());
  return ious;
}

/**
 * Each block will handle one channel of each image
 **/
template <typename T>
__global__
void HorizFlipImagesAndBoxes(
                             const int N,
                             const int C,
                             const int H,
                             const int W,
                             const T* img_in,
                             float* bboxes,
                             const int* offsets,
                             const float p,
                             const float* flip,
                             T* img_out,
                             const bool nhwc) {
  // early return if not flipping
  if (flip[blockIdx.x] < p) return;

  // pointer offset into images
  const int img_offset = blockIdx.x * C * H * W;
  const T* img = &img_in[img_offset];
  T* img_o = &img_out[img_offset];

  // flip bboxes
  auto bbox_offset_begin = offsets[blockIdx.x];
  auto bbox_offset_end   = offsets[blockIdx.x + 1];
  auto num_bboxes = bbox_offset_end - bbox_offset_begin;

  const int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;

  // bboxes in ltrb format, scaled to [0, 1]
  for (int i = thread_idx; i < num_bboxes; i += blockDim.x * blockDim.y) {
    float *bbox = &bboxes[(bbox_offset_begin + thread_idx) * 4];
    // Could do this inplace, but not register constrained
    auto bbox_0 = bbox[0];
    auto bbox_2 = bbox[2];
    bbox[0] = 1. - bbox_2;
    bbox[2] = 1. - bbox_0;
  }

  if (nhwc) {
    // loop over float3 pixels, handle 3 values / thread
    for (int h = threadIdx.y; h < H; h += blockDim.y) {
      for (int w = threadIdx.x; w < W; w += blockDim.x) {
        const T* img_hw = &img[h * W * C + w * C];
        T * img_out_hw = &img_o[h * W * C + (W - 1 - w) * C];

        for (int c = 0; c < C; ++c) {
          img_out_hw[c] = img_hw[c];
        }
      }
    }
  } else {
    // loop over channels
    for (int c = 0; c < C; ++c) {
      const T* img_c = &img[c * H * W];
      T *img_out_c = &img_o[c * H * W];

      // handle tiles of (h, w) at a time
      for (int h = threadIdx.y; h < H; h += blockDim.y) {
        for (int w = threadIdx.x; w < W; w += blockDim.x) {
          const int input_idx = h * W + w;
          const int output_idx = h * W + (W - 1 - w);


          img_out_c[output_idx] = img_c[input_idx];
        }
      }
    }
  }
}

/**
  * Take images and their bboxes, randomly flip on horizontal axis
  * In/Out: img: NCHW tensor of N, C-channel images of constant (H, W)
  * In/Out: bboxes: [N_i, 4] tensor of original bboxes in ltrb format
  * In: bbox_offsets: [N] offset values into bboxes
  * In: p \in [0, 1): probability of flipping each (img, bbox) pair
  * In: nhwc: Tensor in NHWC format
  * ----
  * Note: allocate temp memory, but effectively do this inplace
  */
std::vector<at::Tensor> random_horiz_flip(
                             at::Tensor& img,
                             at::Tensor& bboxes,
                             const at::Tensor& bbox_offsets,
                             const float p,
                             const bool nhwc) {
  // dimensions
  const int N = img.size(0);
  int C, H, W;
  if (nhwc) {
    C = img.size(3);
    H = img.size(1);
    W = img.size(2);

  } else {
    C = img.size(1);
    H = img.size(2);
    W = img.size(3);
  }

  assert(img.is_cuda());
  assert(bboxes.is_cuda());
  assert(bbox_offsets.is_cuda());

  // printf("%d %d %d %d\n", N, C, H, W);
  // Need temp storage of size img
  at::Tensor tmp_img = img.clone();
  at::Tensor flip = at::zeros({N}, at::CUDA(at::kFloat)).uniform_(0., 1.);

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      img.scalar_type(),
      "HorizFlipImagesAndBoxes",
      [&] {
        HorizFlipImagesAndBoxes<scalar_t><<<N, dim3(16, 16), 0, stream.stream()>>>(
          N,
          C,
          H,
          W,
          img.data_ptr<scalar_t>(),
          bboxes.data_ptr<float>(),
          bbox_offsets.data_ptr<int>(),
          p,
          flip.data_ptr<float>(),
          tmp_img.data_ptr<scalar_t>(),
          nhwc);
        THCudaCheck(hipGetLastError());
      });

  // copy tmp_img -> img
  // img = tmp_img;

  return {tmp_img, bboxes};
}
