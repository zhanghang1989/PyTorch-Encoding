#include "hip/hip_runtime.h"
#include <vector>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/Functions.h>
#include <ATen/cuda/HIPContext.h>

#include "common.h"
#include "device_tensor.h"

namespace {

template<typename DType, typename Acctype>
struct KD2Op {
  __device__ KD2Op(DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c,
                   DeviceTensor<DType, 2> std) : X(x), C(c), STD(std) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) 
  {
      DType r = (X[b][i][d] - C[k][d]) / STD[k][d];
      return ScalarConvert<DType, Acctype>::to(r * r);
  }
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
};

template<typename DType, typename Acctype>
__global__ void Encoding_Dist_Forward_kernel (
    DeviceTensor<DType, 3> KD,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  k = blockIdx.x;
  i = blockIdx.y;
  D = X.getSize(2);
  /* main operation */
  KD2Op<DType, Acctype> g(X, C, STD);
  KD[b][i][k] = reduceD<Acctype>(g, b, i, k, D);;
}

template<typename DType, typename Acctype>
struct EncGradXOp {
  __device__ EncGradXOp(
    DeviceTensor<DType, 3> gkd,
    DeviceTensor<DType, 3> x,
    DeviceTensor<DType, 2> c,
    DeviceTensor<DType, 2> std) : GKD(gkd), X(x), C(c), STD(std) {}
    // DeviceTensor<DType, 1> s, S(s)
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(
      2 * GKD[b][i][k] * (X[b][i][d] - C[k][d]) / 
      (STD[k][d] * STD[k][d]));
  }
  DeviceTensor<DType, 3> GKD;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
  // DeviceTensor<DType, 1> S;
};

template<typename DType, typename Acctype>
__global__ void Encoding_GradX_kernel (
    DeviceTensor<DType, 3> GKD,
    DeviceTensor<DType, 3> GX,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD) {
    // DeviceTensor<DType, 1> S
  /* declarations of the variables */
  int b, d, i, K;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  d = blockIdx.x;
  K = C.getSize(0);
  /* main operation */
  EncGradXOp<DType, Acctype> g(GKD, X, C, STD);
  GX[b][i][d] = reduceK<Acctype>(g, b, i, d, K);
}

template<typename DType, typename Acctype>
struct EncGradSTDOp {
  __device__ EncGradSTDOp(
    DeviceTensor<DType, 3> gkd,
    DeviceTensor<DType, 3> x,
    DeviceTensor<DType, 2> c,
    DeviceTensor<DType, 2> std) : GKD(gkd), X(x), C(c), STD(std) {}
    // DeviceTensor<DType, 1> s, S(s)
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(
      -2 * GKD[b][i][k] * (X[b][i][d] - C[k][d]) *
      (X[b][i][d] - C[k][d]) / (STD[k][d] * STD[k][d] * STD[k][d]));
  }
  DeviceTensor<DType, 3> GKD;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
  // DeviceTensor<DType, 1> S;
};

template<typename DType, typename Acctype>
__global__ void Encoding_GradCSTD_kernel (
    DeviceTensor<DType, 3> GKD,
    DeviceTensor<DType, 2> GC,
    DeviceTensor<DType, 2> GSTD,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD) {
  /* declarations of the variables */
  int k, d, B, N;
  /* Get the index and channels */ 
  d = blockIdx.x;
  k = blockIdx.y;
  B = X.getSize(0);
  N = X.getSize(1);
  /* main operation */
  EncGradXOp<DType, Acctype> g1(GKD, X, C, STD);
  EncGradSTDOp<DType, Acctype> g2(GKD, X, C, STD);
  GC[k][d] = -reduceBN<Acctype>(g1, k, d, B, N);
  GSTD[k][d] += reduceBN<Acctype>(g2, k, d, B, N);
}

template<typename DType, typename Acctype>
struct EncGradSTDXOp {
  __device__ EncGradSTDXOp(
    DeviceTensor<DType, 2> gstd,
    DeviceTensor<DType, 3> x,
    DeviceTensor<DType, 2> c,
    DeviceTensor<DType, 2> std) : GSTD(gstd), X(x), C(c), STD(std) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(
      GSTD[k][d] * (X[b][i][d] - C[k][d]) / STD[k][d]);
  }
  DeviceTensor<DType, 2> GSTD;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
};

template<typename DType, typename Acctype>
__global__ void Encoding_GradSTDX_kernel (
    DeviceTensor<DType, 2> GSTD,
    DeviceTensor<DType, 3> GX,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD,
    int N) {
  /* declarations of the variables */
  int b, d, i, K;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  d = blockIdx.x;
  K = C.getSize(0);
  /* main operation */
  EncGradSTDXOp<DType, Acctype> g(GSTD, X, C, STD);
  GX[b][i][d] += reduceK<Acctype>(g, b, i, d, K) / N;
}

template<typename DType, typename Acctype>
struct AggOpV2 {
  __device__ AggOpV2(DeviceTensor<DType, 3> a,
                     DeviceTensor<DType, 3> x,
                     DeviceTensor<DType, 2> c,
                     DeviceTensor<DType, 2> std) : A(a), X(x), C(c), STD(std) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(A[b][i][k] * (X[b][i][d] - C[k][d]) /
                                             STD[k][d]);
  }
  DeviceTensor<DType, 3> A;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
};

template<typename DType, typename Acctype>
__global__ void AggregateV2_Forward_kernel (
    DeviceTensor<DType, 3> E,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD) {
  /* declarations of the variables */
  int b, k, d, N;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  k = blockIdx.y;
  N = X.getSize(1);
  /* main operation */
  AggOpV2<DType, Acctype> g(A, X, C, STD);
  E[b][k][d] = reduceN<Acctype>(g, b, k, d, N);
}

template<typename DType, typename Acctype>
struct AggV2BackOp {
  __device__ AggV2BackOp(DeviceTensor<DType, 3> g,
                         DeviceTensor<DType, 3> x,
                         DeviceTensor<DType, 2> c,
                         DeviceTensor<DType, 2> std) : G(g), X(x), C(c), STD(std) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(G[b][k][d] * (X[b][i][d] - C[k][d]) /
                                             STD[k][d]);
  }
  DeviceTensor<DType, 3> G;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 2> STD;
};

template<typename DType, typename Acctype>
__global__ void AggregateV2_Backward_kernel (
    DeviceTensor<DType, 3> GA,
    DeviceTensor<DType, 3> GE,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 2> STD) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  k = blockIdx.x;
  D = GE.getSize(2);
  /* main operation */
  AggV2BackOp<DType, Acctype> g(GE, X, C, STD);
  GA[b][i][k] = reduceD<Acctype>(g, b, i, k, D);
}

} // namespace

at::Tensor Encoding_Dist_Inference_Forward_CUDA(
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor STD_) {
    // const at::Tensor S_,
  // X \in R^{B, N, D}, C \in R^{K, D}, S \in R^K
  auto KD_ = torch::zeros({X_.size(0), X_.size(1), C_.size(0)}, X_.options());
  // E(x), E(x^2)
  int N = X_.size(0) * X_.size(1);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  // calculate the kernel distance
  AT_DISPATCH_FLOATING_TYPES(X_.type(), "Encoding_Dist_Inference_Forward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> KD = devicetensor<scalar_t, 3>(KD_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    /* kernel function */
    Encoding_Dist_Forward_kernel<scalar_t, scalar_t>
        <<<blocks, threads, 0, stream>>> (KD, X, C, STD);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return KD_;
}

std::vector<at::Tensor> Encoding_Dist_Inference_Backward_CUDA(
    const at::Tensor GKD_,
    const at::Tensor KD_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor STD_) {
  auto GX_ = at::zeros_like(X_);
  auto GC_ = at::zeros_like(C_);
  auto GSTD_ = at::zeros_like(STD_);
  /* kernel function */
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks1(X_.size(2), X_.size(1), X_.size(0));
  dim3 threads1(getNumThreads(C_.size(0)));
  dim3 blocks2(C_.size(1), C_.size(0));
  dim3 threads2(getNumThreads(X_.size(1)));
  int N = X_.size(0) * X_.size(1);
  AT_DISPATCH_FLOATING_TYPES(X_.type(), "Encoding_Dist_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GKD = devicetensor<scalar_t, 3>(GKD_);
    DeviceTensor<scalar_t, 2> GSTD = devicetensor<scalar_t, 2>(GSTD_);
    DeviceTensor<scalar_t, 3> GX = devicetensor<scalar_t, 3>(GX_);
    DeviceTensor<scalar_t, 2> GC = devicetensor<scalar_t, 2>(GC_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    Encoding_GradX_kernel<scalar_t, scalar_t>
      <<<blocks1, threads1, 0, stream>>> (GKD, GX, X, C, STD);
    AT_ASSERT(hipGetLastError() == hipSuccess);
    Encoding_GradCSTD_kernel<scalar_t, scalar_t>
      <<<blocks2, threads2, 0, stream>>> (GKD, GC, GSTD, X, C, STD);
    AT_ASSERT(hipGetLastError() == hipSuccess);
  }));
  return {GX_, GC_, GSTD_};
}

std::vector<at::Tensor> Encoding_Dist_Forward_CUDA(
    const at::Tensor X_,
    const at::Tensor C_,
    double eps) {
    // const at::Tensor S_,
  // X \in R^{B, N, D}, C \in R^{K, D}, S \in R^K
  auto KD_ = torch::zeros({X_.size(0), X_.size(1), C_.size(0)}, X_.options());
  // E(x), E(x^2)
  int N = X_.size(0) * X_.size(1);
  auto SVar_ = (X_.pow(2).sum(0).sum(0).view({1, X_.size(2)}) -
                2 * C_ * X_.sum(0).sum(0).view({1, X_.size(2)})).expand_as(C_) +
               C_.pow(2) * N;
  auto STD_ = at::sqrt(SVar_ / N + eps);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  // calculate the kernel distance
  AT_DISPATCH_FLOATING_TYPES(X_.type(), "Encoding_Dist_Forward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> KD = devicetensor<scalar_t, 3>(KD_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    /* kernel function */
    Encoding_Dist_Forward_kernel<scalar_t, scalar_t>
        <<<blocks, threads, 0, stream>>> (KD, X, C, STD);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {KD_, STD_, SVar_ / (N - 1)};
}

std::vector<at::Tensor> Encoding_Dist_Backward_CUDA(
    const at::Tensor GKD_,
    const at::Tensor GSTD_,
    const at::Tensor KD_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor STD_) {
  auto GX_ = at::zeros_like(X_);
  auto GC_ = at::zeros_like(C_);
  auto GSTD2_ = GSTD_.clone();
  /* kernel function */
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks1(X_.size(2), X_.size(1), X_.size(0));
  dim3 threads1(getNumThreads(C_.size(0)));
  dim3 blocks2(C_.size(1), C_.size(0));
  dim3 threads2(getNumThreads(X_.size(1)));
  int N = X_.size(0) * X_.size(1);
  AT_DISPATCH_FLOATING_TYPES(X_.type(), "Encoding_Dist_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GKD = devicetensor<scalar_t, 3>(GKD_);
    DeviceTensor<scalar_t, 2> GSTD = devicetensor<scalar_t, 2>(GSTD2_);
    DeviceTensor<scalar_t, 3> GX = devicetensor<scalar_t, 3>(GX_);
    DeviceTensor<scalar_t, 2> GC = devicetensor<scalar_t, 2>(GC_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    Encoding_GradX_kernel<scalar_t, scalar_t>
      <<<blocks1, threads1, 0, stream>>> (GKD, GX, X, C, STD);
    AT_ASSERT(hipGetLastError() == hipSuccess);
    Encoding_GradCSTD_kernel<scalar_t, scalar_t>
      <<<blocks2, threads2, 0, stream>>> (GKD, GC, GSTD, X, C, STD);
    AT_ASSERT(hipGetLastError() == hipSuccess);
    Encoding_GradSTDX_kernel<scalar_t, scalar_t>
      <<<blocks1, threads1, 0, stream>>> (GSTD, GX, X, C, STD, N);
    AT_ASSERT(hipGetLastError() == hipSuccess);
  }));
  // d_sigma/d_c
  GC_ = GC_ - GSTD2_ * (X_.mean(0).mean(0) - C_) / STD_;
  return {GX_, GC_};
}

at::Tensor AggregateV2_Forward_CUDA(
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor STD_) {
  /* Device tensors */
  auto E_ = torch::zeros({A_.size(0), C_.size(0), C_.size(1)}, A_.options());
  // auto IS_ = 1.0f / (S_ + eps).sqrt();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(1), C_.size(0), X_.size(0));
  dim3 threads(getNumThreads(X_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Forward_CUDA", ([&] {
    DeviceTensor<scalar_t, 3> E = devicetensor<scalar_t, 3>(E_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    /* kernel function */
    AggregateV2_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>>(E, A, X, C, STD);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return E_;
}

std::vector<at::Tensor> AggregateV2_Backward_CUDA(
    const at::Tensor GE_,
    const at::Tensor E_,
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor STD_) {
  auto gradA_ = at::zeros_like(A_);
  auto gradX_ = at::bmm(A_ , (GE_ / STD_.unsqueeze(0)));
  auto gradC_ = -(A_.sum(1).unsqueeze(2) * GE_ / STD_.unsqueeze(0)).sum(0);
  auto gradSTD_ = -(GE_ * E_).sum(0) / STD_;
  // auto gradS_ = -0.5 * (GE_ * E_).sum(2).sum(0) / S_;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GA = devicetensor<scalar_t, 3>(gradA_);
    DeviceTensor<scalar_t, 3> GE = devicetensor<scalar_t, 3>(GE_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 2> STD = devicetensor<scalar_t, 2>(STD_);
    AggregateV2_Backward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (GA, GE, A, X, C, STD);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {gradA_, gradX_, gradC_, gradSTD_};
}
