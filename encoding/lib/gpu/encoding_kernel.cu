#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <vector>

#include "common.h"
#include "device_tensor.h"

namespace {

template<typename DType, typename Acctype>
struct AggOp {
  __device__ AggOp(DeviceTensor<DType, 3> a,
                   DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c) : A(a), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(A[b][i][k] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> A;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct AggBackOp {
  __device__ AggBackOp(DeviceTensor<DType, 3> g,
                       DeviceTensor<DType, 3> x,
                       DeviceTensor<DType, 2> c) : G(g), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(G[b][k][d] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> G;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct SL2Op {
  __device__ SL2Op(DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c) : X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) 
  {
      DType r = X[b][i][d] - C[k][d];
      return ScalarConvert<DType, Acctype>::to(r * r);
  }
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct SL2GradXOp {
  __device__ SL2GradXOp(
    DeviceTensor<DType, 3> gsl,
    DeviceTensor<DType, 3> x,
    DeviceTensor<DType, 2> c,
    DeviceTensor<DType, 1> s
  ) : GSL(gsl), X(x), C(c), S(s) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) 
  {
    return ScalarConvert<DType, Acctype>::to(
      2 * S[k] * GSL[b][i][k] * (X[b][i][d]-C[k][d]));
  }
  DeviceTensor<DType, 3> GSL;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 1> S;
};


template<typename T, typename Op>
__device__ T reduceN(
    Op op, int b, int k, int d, int N) {
  T sum = 0;
  for (int x = threadIdx.x; x < N; x += blockDim.x) {
      sum += op(b,x,k,d);
  }
  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];

  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
      if (threadIdx.x / WARP_SIZE < 32) {
              shared[threadIdx.x / WARP_SIZE] = sum;
      }
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
      // zero out the other entries in shared
      shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
      sum = warpSum(shared[threadIdx.x]);
      if (threadIdx.x == 0) {
          shared[0] = sum;
      }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template<typename T, typename Op>
__device__ T reduceD(
    Op op, int b, int i, int k, int D) {
  T sum = 0;
  for (int x = threadIdx.x; x < D; x += blockDim.x) {
      sum += op(b,i,k,x);
  }
  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];

  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
      if (threadIdx.x / WARP_SIZE < 32) {
              shared[threadIdx.x / WARP_SIZE] = sum;
      }
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
      // zero out the other entries in shared
      shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
      sum = warpSum(shared[threadIdx.x]);
      if (threadIdx.x == 0) {
          shared[0] = sum;
      }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template<typename T, typename Op>
__device__ T reduceK(
    Op op, int b, int i, int d, int K) {
  T sum = 0;
  for (int x = threadIdx.x; x < K; x += blockDim.x) {
    sum += op(b,i,x,d);
  }
  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];

  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
    if (threadIdx.x / WARP_SIZE < 32) {
            shared[threadIdx.x / WARP_SIZE] = sum;
    }
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
    // zero out the other entries in shared
    shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
    sum = warpSum(shared[threadIdx.x]);
    if (threadIdx.x == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template<typename T, typename Op>
__device__ T reduceBN(
    Op op, 
    int k, int d, int B, int N) {
  T sum = 0;
  for (int batch = 0; batch < B; ++batch) {
    for (int x = threadIdx.x; x < N; x += blockDim.x) {
        sum += op(batch,x,k,d);
    }
  }
  // sum over NumThreads within a warp
  sum = warpSum(sum);
  // 'transpose', and reduce within warp again
  __shared__ T shared[32];

  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
    if (threadIdx.x / WARP_SIZE < 32) {
            shared[threadIdx.x / WARP_SIZE] = sum;
    }
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
    // zero out the other entries in shared
    shared[threadIdx.x] = (T) 0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
    sum = warpSum(shared[threadIdx.x]);
    if (threadIdx.x == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template<typename DType, typename Acctype>
__global__ void Aggregate_Forward_kernel (
    DeviceTensor<DType, 3> E,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, d, N;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  k = blockIdx.y;
  N = X.getSize(1);
  /* main operation */
  AggOp<DType, Acctype> g(A,X,C);
  E[b][k][d] = reduceN<Acctype>(g, b, k, d, N);
}

template<typename DType, typename Acctype>
__global__ void Aggregate_Backward_kernel (
    DeviceTensor<DType, 3> GA,
    DeviceTensor<DType, 3> GE,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  k = blockIdx.x;
  D = GE.getSize(2);
  /* main operation */
  AggBackOp<DType, Acctype> g(GE,X,C);
  GA[b][i][k] = reduceD<Acctype>(g, b, i, k, D);
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_Forward_kernel (
    DeviceTensor<DType, 3> SL,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  k = blockIdx.x;
  i = blockIdx.y;
  D = X.getSize(2);
  /* main operation */
  SL2Op<DType, Acctype> g(X,C);
  SL[b][i][k] = S[k] * reduceD<Acctype>(g,b,i,k,D);;
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_GradX_kernel (
    DeviceTensor<DType, 3> GSL,
    DeviceTensor<DType, 3> GX,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int b, d, i, K;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  i = blockIdx.y;
  K = C.getSize(0);
  /* main operation */
  SL2GradXOp<DType, Acctype> g(GSL,X,C,S);
  GX[b][i][d] = reduceK<Acctype>(g,b,i,d,K);
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_GradC_kernel (
    DeviceTensor<DType, 3> GSL,
    DeviceTensor<DType, 2> GC,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int k, d, B, N;
  /* Get the index and channels */ 
  d = blockIdx.x;
  k = blockIdx.y;
  B = X.getSize(0);
  N = X.getSize(1);
  /* main operation */
  SL2GradXOp<DType, Acctype> g(GSL,X,C,S);
  GC[k][d] = - reduceBN<Acctype>(g, k, d, B, N);
}

}// namespace

at::Tensor Aggregate_Forward_CUDA(
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  /* Device tensors */
  auto E_ = A_.type().tensor({A_.size(0), C_.size(0), C_.size(1)}).zero_(); 
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(1), C_.size(0), X_.size(0));
  dim3 threads(getNumThreads(X_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Forward_CUDA", ([&] {
    DeviceTensor<scalar_t, 3> E = devicetensor<scalar_t, 3>(E_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    /* kernel function */
    Aggregate_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>>(E, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return E_;
}

std::vector<at::Tensor> Aggregate_Backward_CUDA(
    const at::Tensor GE_,
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  auto gradA_ = at::zeros_like(A_);
  auto gradX_ = at::bmm(A_, GE_);
  auto gradC_ = (-GE_ * A_.sum(1).unsqueeze(2)).sum(0);
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GA = devicetensor<scalar_t, 3>(gradA_);
    DeviceTensor<scalar_t, 3> GE = devicetensor<scalar_t, 3>(GE_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    Aggregate_Backward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (GA, GE, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {gradA_, gradX_, gradC_};
}

at::Tensor ScaledL2_Forward_CUDA(
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor S_) {
  auto SL_ = X_.type().tensor({X_.size(0), X_.size(1), C_.size(0)}).zero_();
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(X_.type(), "ScaledL2_Forward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> SL = devicetensor<scalar_t, 3>(SL_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 1> S = devicetensor<scalar_t, 1>(S_);
    /* kernel function */
    ScaledL2_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (SL, X, C, S);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return SL_;
}

std::vector<at::Tensor> ScaledL2_Backward_CUDA(
    const at::Tensor GSL_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor S_,
    const at::Tensor SL_) {
  auto GX_ = at::zeros_like(X_);
  auto GC_ = at::zeros_like(C_);
  /* kernel function */
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks1(X_.size(2), X_.size(1), X_.size(0));
  dim3 threads1(getNumThreads(C_.size(0)));
  dim3 blocks2(C_.size(1), C_.size(0));
  dim3 threads2(getNumThreads(X_.size(1)));
  //std::vector<int> size{ 1, 1, K};
  //auto GS_ = GSL_ * (SL_ / at::_unsafe_view(S_, size))
  auto GS_ = (GSL_ * (SL_ / S_.view({1, 1, C_.size(0)}))).sum(0).sum(0);
  AT_DISPATCH_FLOATING_TYPES(X_.type(), "ScaledL2_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GSL = devicetensor<scalar_t, 3>(GSL_);
    DeviceTensor<scalar_t, 3> GX = devicetensor<scalar_t, 3>(GX_);
    DeviceTensor<scalar_t, 2> GC = devicetensor<scalar_t, 2>(GC_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 1> S = devicetensor<scalar_t, 1>(S_);
    ScaledL2_GradX_kernel<scalar_t, scalar_t>
      <<<blocks1, threads1, 0, stream>>> (GSL, GX, X, C, S);
    AT_ASSERT(hipGetLastError() == hipSuccess);
    ScaledL2_GradC_kernel<scalar_t, scalar_t>
      <<<blocks2, threads2, 0, stream>>> (GSL, GC, X, C, S);
    AT_ASSERT(hipGetLastError() == hipSuccess);
  }));
  return {GX_, GC_, GS_};
}
