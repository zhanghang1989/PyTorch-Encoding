#include "hip/hip_runtime.h"
#include <vector>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "common.h"
#include "device_tensor.h"

namespace {

template<typename DType, typename Acctype>
struct AggOp {
  __device__ AggOp(DeviceTensor<DType, 3> a,
                   DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c) : A(a), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(A[b][i][k] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> A;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct AggBackOp {
  __device__ AggBackOp(DeviceTensor<DType, 3> g,
                       DeviceTensor<DType, 3> x,
                       DeviceTensor<DType, 2> c) : G(g), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(G[b][k][d] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> G;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct SL2Op {
  __device__ SL2Op(DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c) : X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) 
  {
      DType r = X[b][i][d] - C[k][d];
      return ScalarConvert<DType, Acctype>::to(r * r);
  }
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct SL2GradXOp {
  __device__ SL2GradXOp(
    DeviceTensor<DType, 3> gsl,
    DeviceTensor<DType, 3> x,
    DeviceTensor<DType, 2> c,
    DeviceTensor<DType, 1> s
  ) : GSL(gsl), X(x), C(c), S(s) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) 
  {
    return ScalarConvert<DType, Acctype>::to(
      2 * S[k] * GSL[b][i][k] * (X[b][i][d]-C[k][d]));
  }
  DeviceTensor<DType, 3> GSL;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
  DeviceTensor<DType, 1> S;
};

template<typename DType, typename Acctype>
__global__ void Aggregate_Forward_kernel (
    DeviceTensor<DType, 3> E,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, d, N;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  k = blockIdx.y;
  N = X.getSize(1);
  /* main operation */
  AggOp<DType, Acctype> g(A, X, C);
  E[b][k][d] = reduceN<Acctype>(g, b, k, d, N);
}

template<typename DType, typename Acctype>
__global__ void Aggregate_Backward_kernel (
    DeviceTensor<DType, 3> GA,
    DeviceTensor<DType, 3> GE,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  k = blockIdx.x;
  D = GE.getSize(2);
  /* main operation */
  AggBackOp<DType, Acctype> g(GE, X, C);
  GA[b][i][k] = reduceD<Acctype>(g, b, i, k, D);
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_Forward_kernel (
    DeviceTensor<DType, 3> SL,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  k = blockIdx.x;
  i = blockIdx.y;
  D = X.getSize(2);
  /* main operation */
  SL2Op<DType, Acctype> g(X,C);
  SL[b][i][k] = S[k] * reduceD<Acctype>(g,b,i,k,D);;
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_GradX_kernel (
    DeviceTensor<DType, 3> GSL,
    DeviceTensor<DType, 3> GX,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int b, d, i, K;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  i = blockIdx.y;
  K = C.getSize(0);
  /* main operation */
  SL2GradXOp<DType, Acctype> g(GSL,X,C,S);
  GX[b][i][d] = reduceK<Acctype>(g,b,i,d,K);
}

template<typename DType, typename Acctype>
__global__ void ScaledL2_GradC_kernel (
    DeviceTensor<DType, 3> GSL,
    DeviceTensor<DType, 2> GC,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C,
    DeviceTensor<DType, 1> S) {
  /* declarations of the variables */
  int k, d, B, N;
  /* Get the index and channels */ 
  d = blockIdx.x;
  k = blockIdx.y;
  B = X.getSize(0);
  N = X.getSize(1);
  /* main operation */
  SL2GradXOp<DType, Acctype> g(GSL,X,C,S);
  GC[k][d] = - reduceBN<Acctype>(g, k, d, B, N);
}

}// namespace

at::Tensor Aggregate_Forward_CUDA(
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  /* Device tensors */
  auto E_ = torch::zeros({A_.size(0), C_.size(0), C_.size(1)}, A_.options());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(1), C_.size(0), X_.size(0));
  dim3 threads(getNumThreads(X_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(A_.scalar_type(), "Aggregate_Forward_CUDA", ([&] {
    DeviceTensor<scalar_t, 3> E = devicetensor<scalar_t, 3>(E_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    /* kernel function */
    Aggregate_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>>(E, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return E_;
}

std::vector<at::Tensor> Aggregate_Backward_CUDA(
    const at::Tensor GE_,
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  auto gradA_ = at::zeros_like(A_);
  auto gradX_ = at::bmm(A_, GE_);
  auto gradC_ = (-GE_ * A_.sum(1).unsqueeze(2)).sum(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  AT_DISPATCH_FLOATING_TYPES(A_.scalar_type(), "Aggregate_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GA = devicetensor<scalar_t, 3>(gradA_);
    DeviceTensor<scalar_t, 3> GE = devicetensor<scalar_t, 3>(GE_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    Aggregate_Backward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (GA, GE, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {gradA_, gradX_, gradC_};
}

at::Tensor ScaledL2_Forward_CUDA(
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor S_) {
  auto SL_ = torch::zeros({X_.size(0), X_.size(1), C_.size(0)}, X_.options());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(X_.scalar_type(), "ScaledL2_Forward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> SL = devicetensor<scalar_t, 3>(SL_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 1> S = devicetensor<scalar_t, 1>(S_);
    /* kernel function */
    ScaledL2_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (SL, X, C, S);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return SL_;
}

std::vector<at::Tensor> ScaledL2_Backward_CUDA(
    const at::Tensor GSL_,
    const at::Tensor X_,
    const at::Tensor C_,
    const at::Tensor S_,
    const at::Tensor SL_) {
  auto GX_ = at::zeros_like(X_);
  auto GC_ = at::zeros_like(C_);
  /* kernel function */
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 blocks1(X_.size(2), X_.size(1), X_.size(0));
  dim3 threads1(getNumThreads(C_.size(0)));
  dim3 blocks2(C_.size(1), C_.size(0));
  dim3 threads2(getNumThreads(X_.size(1)));
  auto GS_ = (GSL_ * (SL_ / S_.view({1, 1, C_.size(0)}))).sum(0).sum(0);
  AT_DISPATCH_FLOATING_TYPES(X_.scalar_type(), "ScaledL2_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GSL = devicetensor<scalar_t, 3>(GSL_);
    DeviceTensor<scalar_t, 3> GX = devicetensor<scalar_t, 3>(GX_);
    DeviceTensor<scalar_t, 2> GC = devicetensor<scalar_t, 2>(GC_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    DeviceTensor<scalar_t, 1> S = devicetensor<scalar_t, 1>(S_);
    ScaledL2_GradX_kernel<scalar_t, scalar_t>
      <<<blocks1, threads1, 0, stream>>> (GSL, GX, X, C, S);
    AT_ASSERT(hipGetLastError() == hipSuccess);
    ScaledL2_GradC_kernel<scalar_t, scalar_t>
      <<<blocks2, threads2, 0, stream>>> (GSL, GC, X, C, S);
    AT_ASSERT(hipGetLastError() == hipSuccess);
  }));
  return {GX_, GC_, GS_};
}
