#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <vector>

#include "common.h"
#include "device_tensor.h"

namespace {

template <typename DType, typename Acctype, typename DeviceTensor3>
struct GradOp {
  __device__ GradOp(Acctype m, const DeviceTensor3 i, const DeviceTensor3 g)
    : mean(m), input(i), gradOutput(g) {}
  __device__ __forceinline__ Float2<DType, Acctype> operator()(int batch, int plane, int n) {
    DType g = gradOutput[batch][plane][n];
    DType c = ScalarConvert<Acctype, DType>::to(input[batch][plane][n] - mean);
    return Float2<DType, Acctype>(g, g * c);
  }
  const Acctype mean;
  const DeviceTensor3 input;
  const DeviceTensor3 gradOutput;
};

template <typename DType, typename Acctype>
struct SumOp {
  __device__ SumOp(DeviceTensor<DType, 3> i) : input(i){}
  __device__ __forceinline__ Float2<DType, Acctype> operator()(int batch, int plane, int n) {
    DType g = input[batch][plane][n];
    return Float2<DType, Acctype>(g, g * g);
  }
  DType mean;
  DeviceTensor<DType, 3> input;
};

// Sum across (batch, x/y/z) applying Op() pointwise
template<typename T, typename Op, typename DeviceTensor3>
__device__ T reduce(Op op, DeviceTensor3 tensor, int plane) {
  T sum = (T)0;
  for (int batch = 0; batch < tensor.getSize(0); ++batch) {
    for (int x = threadIdx.x; x < tensor.getSize(2); x += blockDim.x) {
      sum += op(batch, plane, x);
    }
  }

  // sum over NumThreads within a warp
  sum = warpSum(sum);

  // 'transpose', and reduce within warp again
  __shared__ T shared[32];
  __syncthreads();
  if (threadIdx.x % WARP_SIZE == 0) {
    shared[threadIdx.x / WARP_SIZE] = sum;
  }
  if (threadIdx.x >= blockDim.x / WARP_SIZE && threadIdx.x < WARP_SIZE) {
    // zero out the other entries in shared
    shared[threadIdx.x] = (T)0;
  }
  __syncthreads();
  if (threadIdx.x / WARP_SIZE == 0) {
    sum = warpSum(shared[threadIdx.x]);
    if (threadIdx.x == 0) {
      shared[0] = sum;
    }
  }
  __syncthreads();

  // Everyone picks it up, should be broadcast into the whole gradInput
  return shared[0];
}

template <typename DType>
__global__ void BatchNorm_Forward_kernel (
  DeviceTensor<DType, 3> output,
  DeviceTensor<DType, 3> input,
  DeviceTensor<DType, 1> mean,
  DeviceTensor<DType, 1> std,
  DeviceTensor<DType, 1> gamma,
  DeviceTensor<DType, 1> beta) {
  int c = blockIdx.x;
  /* main operation */ 
  for (int b = 0; b < input.getSize(0); ++b) {
    for (int x = threadIdx.x; x < input.getSize(2); x += blockDim.x) {
      DType inp = input[b][c][x];
      output[b][c][x] = gamma[c] * (inp - mean[c]) /
        std[c] + beta[c];
    }
  }
}

template <typename DType>
__global__ void BatchNorm_Backward_kernel (
    DeviceTensor<DType, 3> gradoutput,
    DeviceTensor<DType, 3> input,
    DeviceTensor<DType, 3> gradinput,
    DeviceTensor<DType, 1> gradgamma,
    DeviceTensor<DType, 1> gradbeta,
    DeviceTensor<DType, 1> mean,
    DeviceTensor<DType, 1> std,
    DeviceTensor<DType, 1> gamma,
    DeviceTensor<DType, 1> beta,
    DeviceTensor<DType, 1> gradMean, 
    DeviceTensor<DType, 1> gradStd,
    bool train) {
  /* declarations of the variables */
  /* Get the index and channels */ 
  int c = blockIdx.x; 
  /* main operation */ 
  GradOp<DType, DType, DeviceTensor<DType, 3>> g(mean[c], input, gradoutput);
  Float2<DType, DType> res = reduce<Float2<DType, DType>,
    GradOp<DType, DType, DeviceTensor<DType, 3>>,
    DeviceTensor<DType, 3>>(g, gradoutput, c);
  DType gradOutputSum = res.v1;
  DType dotP = res.v2;
  DType invstd = DType(1.0) / std[c];
  DType gradScale = invstd * gamma[c];
  if (train && threadIdx.x == 0) {
    gradMean[c] = - gradOutputSum * gamma[c] * invstd;
    gradStd[c]  = - dotP * gamma[c] * invstd * invstd;
  }
  if (gradinput.numElements() > 0) {
    for (int batch = 0; batch < gradoutput.getSize(0); ++batch) {
      for (int x = threadIdx.x; x < gradoutput.getSize(2); x += blockDim.x) {
        gradinput[batch][c][x] = gradoutput[batch][c][x] * gradScale;
      }
    }
  }
  if (gradgamma.numElements() > 0) {
    if (threadIdx.x == 0) {
      gradgamma[c] += dotP * invstd;
    }
  }
  if (gradbeta.numElements() > 0) {
    if (threadIdx.x == 0) {
      gradbeta[c] += gradOutputSum;
    }
  }
}


template <typename DType>
__global__ void Sum_Square_Forward_kernel (
    DeviceTensor<DType, 3> input,
    DeviceTensor<DType, 1> sum,
    DeviceTensor<DType, 1> square) {
  int c = blockIdx.x;
  /* main operation */ 
  SumOp<DType, DType> g(input);
  Float2<DType, DType> res = reduce<Float2<DType, DType>,
    SumOp<DType, DType>, DeviceTensor<DType, 3>>(g, input, c);
  DType xsum = res.v1;
  DType xsquare = res.v2;
  if (threadIdx.x == 0) {
    sum[c] = xsum;
    square[c] = xsquare;
  }
}

template <typename DType>
__global__ void Sum_Square_Backward_kernel (
  DeviceTensor<DType, 3> gradInput,
  DeviceTensor<DType, 3> input,
  DeviceTensor<DType, 1> gradSum,
  DeviceTensor<DType, 1> gradSquare) {
  int c = blockIdx.x;
  /* main operation */ 
  for (int batch = 0; batch < gradInput.getSize(0); ++batch) {
    for (int x = threadIdx.x; x < gradInput.getSize(2); x += blockDim.x)
    {
      gradInput[batch][c][x] = gradSum[c] + 2 * gradSquare[c] *
          input[batch][c][x];
    }
  }   
}

} // namespcae

at::Tensor BatchNorm_Forward_CUDA(
    const at::Tensor input_, 
    const at::Tensor mean_,
    const at::Tensor std_,
    const at::Tensor gamma_,
    const at::Tensor beta_) {
  auto output_ = at::zeros_like(input_);
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks(input_.size(1));
  dim3 threads(getNumThreads(input_.size(2)));
  AT_DISPATCH_FLOATING_TYPES(input_.type(), "BatchNorm_Forward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> output = devicetensor<scalar_t, 3>(output_);
    DeviceTensor<scalar_t, 3> input = devicetensor<scalar_t, 3>(input_);
    DeviceTensor<scalar_t, 1> mean = devicetensor<scalar_t, 1>(mean_);
    DeviceTensor<scalar_t, 1> std = devicetensor<scalar_t, 1>(std_);
    DeviceTensor<scalar_t, 1> gamma = devicetensor<scalar_t, 1>(gamma_);
    DeviceTensor<scalar_t, 1> beta = devicetensor<scalar_t, 1>(beta_);
    /* kernel function */
    BatchNorm_Forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        output, input, mean, std, gamma, beta);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return output_;
}

std::vector<at::Tensor> BatchNorm_Backward_CUDA(
    const at::Tensor gradoutput_,
    const at::Tensor input_,
    const at::Tensor mean_, 
    const at::Tensor std_,
    const at::Tensor gamma_,
    const at::Tensor beta_, 
    bool train) {
  /* outputs*/
  at::Tensor gradinput_ = at::zeros_like(input_);
  at::Tensor gradgamma_ = at::zeros_like(gamma_);
  at::Tensor gradbeta_ = at::zeros_like(beta_);
  at::Tensor gradMean_ = at::zeros_like(mean_);
  at::Tensor gradStd_ = at::zeros_like(std_);
  /* cuda utils*/
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks(input_.size(1));
  dim3 threads(getNumThreads(input_.size(2)));
  AT_DISPATCH_FLOATING_TYPES(input_.type(), "BatchNorm_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> gradoutput = devicetensor<scalar_t, 3>(gradoutput_);
    DeviceTensor<scalar_t, 3> input = devicetensor<scalar_t, 3>(input_);
    DeviceTensor<scalar_t, 3> gradinput = devicetensor<scalar_t, 3>(gradinput_);
    DeviceTensor<scalar_t, 1> gradgamma = devicetensor<scalar_t, 1>(gradgamma_);
    DeviceTensor<scalar_t, 1> gradbeta = devicetensor<scalar_t, 1>(gradbeta_);
    DeviceTensor<scalar_t, 1> mean = devicetensor<scalar_t, 1>(mean_);
    DeviceTensor<scalar_t, 1> std = devicetensor<scalar_t, 1>(std_);
    DeviceTensor<scalar_t, 1> gamma = devicetensor<scalar_t, 1>(gamma_);
    DeviceTensor<scalar_t, 1> beta = devicetensor<scalar_t, 1>(beta_);
    DeviceTensor<scalar_t, 1> gradMean = devicetensor<scalar_t, 1>(gradMean_);
    DeviceTensor<scalar_t, 1> gradStd = devicetensor<scalar_t, 1>(gradStd_);
    /* kernel function */
    BatchNorm_Backward_kernel<scalar_t>
      <<<blocks, threads, 0, stream>>>(
      gradoutput, input, gradinput, gradgamma, gradbeta, mean, std, 
      gamma, beta, gradMean, gradStd, train);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {gradinput_, gradMean_, gradStd_, gradgamma_, gradbeta_};
}

std::vector<at::Tensor> Sum_Square_Forward_CUDA(
    const at::Tensor input_) {
  /* outputs */
  at::Tensor sum_ = input_.type().tensor({input_.size(1)}).zero_();
  at::Tensor square_ = input_.type().tensor({input_.size(1)}).zero_();
  /* cuda utils*/
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks(input_.size(1));
  dim3 threads(getNumThreads(input_.size(2)));
  AT_DISPATCH_FLOATING_TYPES(input_.type(), "BatchNorm_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> input = devicetensor<scalar_t, 3>(input_);
    DeviceTensor<scalar_t, 1> sum = devicetensor<scalar_t, 1>(sum_);
    DeviceTensor<scalar_t, 1> square = devicetensor<scalar_t, 1>(square_);
    /* kernel function */
    Sum_Square_Forward_kernel<scalar_t>
      <<<blocks, threads, 0, stream>>>(input, sum, square);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {sum_, square_};
}

at::Tensor Sum_Square_Backward_CUDA(
    const at::Tensor input_,
    const at::Tensor gradSum_,
    const at::Tensor gradSquare_) {
  /* outputs */
  at::Tensor gradInput_ = at::zeros_like(input_);
  /* cuda utils*/
  hipStream_t stream = at::globalContext().getCurrentCUDAStream();
  dim3 blocks(input_.size(1));
  dim3 threads(getNumThreads(input_.size(2)));
  AT_DISPATCH_FLOATING_TYPES(input_.type(), "BatchNorm_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> gradInput = devicetensor<scalar_t, 3>(gradInput_);
    DeviceTensor<scalar_t, 3> input = devicetensor<scalar_t, 3>(input_);
    DeviceTensor<scalar_t, 1> gradSum = devicetensor<scalar_t, 1>(gradSum_);
    DeviceTensor<scalar_t, 1> gradSquare =devicetensor<scalar_t, 1>(gradSquare_);
    /* kernel function */
    Sum_Square_Backward_kernel<scalar_t>
      <<<blocks, threads, 0, stream>>>(gradInput, input, gradSum, gradSquare);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return gradInput_;
}
