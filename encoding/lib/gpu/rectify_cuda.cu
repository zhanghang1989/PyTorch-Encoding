#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/div_rtn.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>


template <typename dest_t, typename src_t>
static inline dest_t safe_downcast(src_t v)
{
  TORCH_CHECK(std::numeric_limits<dest_t>::min() <= v && v <= std::numeric_limits<dest_t>::max(),
              "integer out of range");

  return static_cast<dest_t>(v);
}

template<typename T>
static inline T pooling_output_shape_pad_lr(
        T inputSize, T kernelSize, T pad_l, T pad_r, T stride, T dilation,
        bool ceil_mode) {
    T outputSize = div_rtn<T>(
        inputSize + pad_l + pad_r - dilation * (kernelSize - 1) - 1 +
        (ceil_mode ? stride - 1 : 0), stride) + 1;
    if (pad_l) {
        // ensure that the last pooling starts inside the image
        // needed to avoid problems in ceil mode
        if ((outputSize - 1) * stride >= inputSize + pad_l)
          --outputSize;
    }
    return outputSize;
}

template<typename T>
static inline T pooling_output_shape(
      T inputSize, T kernelSize, T pad, T stride, T dilation, bool ceil_mode) {
    return pooling_output_shape_pad_lr(
        inputSize, kernelSize, pad, pad, stride, dilation, ceil_mode);
}

static inline void pool2d_shape_check(
  const at::Tensor& input,
  int kH, int kW, int dH, int dW, int padH, int padW, int dilationH, int dilationW,
  int64_t nInputPlane,
  int64_t inputHeight, int64_t inputWidth,
  int64_t outputHeight, int64_t outputWidth)
{
  const int64_t ndim = input.ndimension();
  const int64_t nOutputPlane = nInputPlane;

  TORCH_CHECK(kW > 0 && kH > 0,
              "kernel size should be greater than zero, but got ",
              "kH: ", kH, " kW: ", kW);
  TORCH_CHECK(dW > 0 && dH > 0,
              "stride should be greater than zero, but got "
              "dH: ", dH, " dW: ", dW);
  TORCH_CHECK(dilationH > 0 && dilationW > 0,
              "dilation should be greater than zero, but got ",
              "dilationH: ", dilationH, " dilationW: ", dilationW);

  TORCH_CHECK(input.numel() > 0 && (ndim == 3 || ndim == 4),
              "non-empty 3D or 4D input tensor expected but got ndim: ", ndim);
  //TORCH_CHECK(kW/2 >= padW && kH/2 >= padH,
  //            "pad should be smaller than half of kernel size, but got ",
  //            "padW = ", padW, ", padH = ", padH, ", kW = ", kW, ", kH = ", kH);

  TORCH_CHECK(outputWidth >= 1 && outputHeight >= 1,
              "Given input size: (",
              nInputPlane, "x", inputHeight, "x", inputWidth, "). ",
              "Calculated output size: (",
              nOutputPlane, "x", outputHeight, "x", outputWidth, "). ",
              "Output size is too small");
}

template <typename scalar_t, typename accscalar_t>
__global__ void conv_rectify_cuda_frame(
    const int nthreads,
    //const scalar_t* const bottom_data,
    const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    scalar_t* const top_data,
    bool average_mode) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    //const int c = (index / pooled_width / pooled_height) % channels;
    //const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = ((kernel_h - 1) / dilation_h + 1) * ((kernel_w - 1) / dilation_w + 1);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    accscalar_t mul_factor;
    int hcount = int(((hend - hstart) - 1) / dilation_h + 1);
    int wcount = int(((wend - wstart) - 1) / dilation_w + 1);
    if (average_mode) {
      mul_factor = accscalar_t(1.0) / (hcount * wcount);
    }
    else {
      mul_factor = accscalar_t(1.0) * pool_size / (hcount * wcount);
    }
    top_data[index] = ScalarConvert<accscalar_t, scalar_t>::to(top_data[index] * mul_factor);
  }
}

void conv_rectify_cuda_tempalte(
  at::Tensor& output,
  const at::Tensor& input_,
  at::IntArrayRef kernel_size,
  at::IntArrayRef stride,
  at::IntArrayRef padding,
  at::IntArrayRef dilation,
  bool average)
{
  //at::TensorArg output_arg{ output, "output", 1 };
  //at::TensorArg input_arg{ input_, "input_", 2 };

  //checkAllSameGPU("rectify_out_cuda", {output_arg, input_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK(kernel_size.size() == 1 || kernel_size.size() == 2,
    "rectify: kernel_size must either be a single int, or a tuple of two ints");
  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  TORCH_CHECK(stride.empty() || stride.size() == 1 || stride.size() == 2,
    "rectify: stride must either be omitted, a single int, or a tuple of two ints");
  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW :
                 stride.size() == 1 ? dH : safe_downcast<int, int64_t>(stride[1]);

  TORCH_CHECK(padding.size() == 1 || padding.size() == 2,
    "rectify: padding must either be a single int, or a tuple of two ints");
  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  TORCH_CHECK(dilation.size() == 1 || dilation.size() == 2,
    "rectify: dilation must either be a single int, or a tuple of two ints");
  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int64_t nbatch = input_.ndimension() == 4 ? input_.size(-4) : 1;
  const int64_t nInputPlane = input_.size(-3);
  const int64_t inputHeight = input_.size(-2);
  const int64_t inputWidth = input_.size(-1);

  //const int64_t outputHeight = pooling_output_shape<int64_t>(inputHeight, kH, padH, dH, dilationH, false);
  //const int64_t outputWidth = pooling_output_shape<int64_t>(inputWidth, kW, padW, dW, dilationW, false);
  const int64_t outputHeight = output.size(-2);
  const int64_t outputWidth = output.size(-1);

  pool2d_shape_check(
    input_,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    nInputPlane,
    inputHeight, inputWidth,
    outputHeight, outputWidth);

  at::Tensor input = input_.contiguous();

  //output.resize_({nbatch, nInputPlane, outputHeight, outputWidth});

  const int32_t count = safe_downcast<int32_t, int64_t>(output.numel());
  const uint32_t  num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  const uint32_t num_blocks = at::cuda::ATenCeilDiv<uint32_t>(count, num_threads);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "conv_rectify_cuda_frame", ([&] {
        //using accscalar_t = acc_type<scalar_t, true>;
        scalar_t *output_data = output.data_ptr<scalar_t>();
        conv_rectify_cuda_frame<scalar_t, scalar_t>
            <<<num_blocks, num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
                nbatch,
                nInputPlane,
                inputHeight, inputWidth,
                outputHeight, outputWidth,
                kH, kW,
                dH, dW,
                padH, padW,
                dilationH, dilationW,
                output_data,
                average);
  }));


  AT_CUDA_CHECK(hipGetLastError());
}

void CONV_RECTIFY_CUDA(
  at::Tensor& output,
  const at::Tensor& input,
  at::IntArrayRef kernel_size,
  at::IntArrayRef stride,
  at::IntArrayRef padding,
  at::IntArrayRef dilation,
  bool average) {
  //at::Tensor output = at::empty({0}, input.options());
  conv_rectify_cuda_tempalte(
    output,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    average);
}

