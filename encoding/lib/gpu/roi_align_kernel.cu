#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {

#define CUDA_1D_KERNEL_LOOP(i, n)                                 \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

// The number of cuda threads to use. 512 is used for backward compatibility
constexpr int ROI_CUDA_NUM_THREADS = 512;

// The maximum number of blocks to use in the default kernel call.
constexpr int ROI_MAXIMUM_NUM_BLOCKS = 4096;

/**
 * @brief Compute the number of blocks needed to run N threads.
 */
inline int ROI_GET_BLOCKS(const int N) {
  return std::max(
      std::min(
          (N + ROI_CUDA_NUM_THREADS - 1) / ROI_CUDA_NUM_THREADS,
          ROI_MAXIMUM_NUM_BLOCKS),
      // Use at least 1 block, since CUDA does not allow empty block
      1);
}

template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = static_cast<int>(y);
  int x_low = static_cast<int>(x);
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForwardKernel(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {  // e.g., iy = 0, 1
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}


template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    T y,
    T x,
    T* w1,
    T* w2,
    T* w3,
    T* w4,
    int* x_low,
    int* x_high,
    int* y_low,
    int* y_high,
    const int /*index*/ /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    *w1 = *w2 = *w3 = *w4 = 0.;
    *x_low = *x_high = *y_low = *y_high = -1;
    return;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  *y_low = static_cast<int>(y);
  *x_low = static_cast<int>(x);

  if (*y_low >= height - 1) {
    *y_high = *y_low = height - 1;
    y = (T)*y_low;
  } else {
    *y_high = *y_low + 1;
  }

  if (*x_low >= width - 1) {
    *x_high = *x_low = width - 1;
    x = (T)*x_low;
  } else {
    *x_high = *x_low + 1;
  }

  T ly = y - *y_low;
  T lx = x - *x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  *w1 = hy * hx, *w2 = hy * lx, *w3 = ly * hx, *w4 = ly * lx;

  return;
}

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__ float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <>
inline __device__ double gpu_atomic_add(const double val, double* address) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                    __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);
  return val;
}

template <typename T>
__global__ void RoIAlignBackwardKernel(
    const int nthreads,
    const T* top_diff,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height);  // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) {  // e.g., iy = 0, 1
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h);  // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height,
            width,
            y,
            x,
            &w1,
            &w2,
            &w3,
            &w4,
            &x_low,
            &x_high,
            &y_low,
            &y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          /*
          atomicAdd(
              offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(
              offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
          */
          gpu_atomic_add(
              static_cast<T>(g1), offset_bottom_diff + y_low * width + x_low);
          gpu_atomic_add(
              static_cast<T>(g2), offset_bottom_diff + y_low * width + x_high);
          gpu_atomic_add(
              static_cast<T>(g3), offset_bottom_diff + y_high * width + x_low);
          gpu_atomic_add(
              static_cast<T>(g4), offset_bottom_diff + y_high * width + x_high);
        }  // if
      }  // ix
    }  // iy
  }  // CUDA_1D_KERNEL_LOOP
}  // RoIAlignBackward
} // namespace


at::Tensor ROIAlign_Forward_CUDA(
    const at::Tensor input,
    const at::Tensor rois,
    int64_t pooled_height,
    int64_t pooled_width,
    double spatial_scale,
    int64_t sampling_ratio) {

  AT_ASSERT(input.is_contiguous());
  AT_ASSERT(rois.is_contiguous());
  AT_ASSERT(input.ndimension() == 4);
  AT_ASSERT(rois.ndimension() == 2);
  AT_ASSERT(rois.size(1) == 5);

  auto proposals = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  // Output Tensor is (num_rois, C, pooled_height, pooled_width)
  auto output = torch::zeros({proposals, channels, pooled_height, pooled_width}, input.options());

  auto count = output.numel();
  
  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "ROIAlign_Forward_CUDA", ([&] {
    RoIAlignForwardKernel<scalar_t>
      <<<ROI_GET_BLOCKS(count),
         ROI_CUDA_NUM_THREADS,
         0,
         at::cuda::getCurrentCUDAStream()>>>(
          count,
          input.data_ptr<scalar_t>(),
          static_cast<scalar_t>(spatial_scale),
          channels,
          height,
          width,
          pooled_height,
          pooled_width,
          sampling_ratio,
          rois.data_ptr<scalar_t>(),
          output.data_ptr<scalar_t>());
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return output;
}

at::Tensor ROIAlign_Backward_CUDA(
    const at::Tensor rois,
    const at::Tensor grad_output,
    int64_t b_size,
    int64_t channels,
    int64_t height,
    int64_t width,
    int64_t pooled_height,
    int64_t pooled_width,
    double spatial_scale,
    int64_t sampling_ratio) {

  AT_ASSERT(rois.is_contiguous());
  AT_ASSERT(rois.ndimension() == 2);
  AT_ASSERT(rois.size(1) == 5);

  auto roi_cols = rois.size(1);
  AT_ASSERT(roi_cols == 4 || roi_cols == 5);

  // Output Tensor is (num_rois, C, pooled_height, pooled_width)
  // gradient wrt input features
  auto grad_in = torch::zeros({b_size, channels, height, width}, rois.options());
  auto num_rois = rois.size(0);
  auto count = grad_output.numel();

  AT_DISPATCH_FLOATING_TYPES(rois.scalar_type(), "ROIAlign_Backward_CUDA", ([&] {
    RoIAlignBackwardKernel<scalar_t>
      <<<ROI_GET_BLOCKS(count),
         ROI_CUDA_NUM_THREADS,
         0,
         at::cuda::getCurrentCUDAStream()>>>(
          count,
          grad_output.data_ptr<scalar_t>(),
          num_rois,
          static_cast<scalar_t>(spatial_scale),
          channels,
          height,
          width,
          pooled_height,
          pooled_width,
          sampling_ratio,
          grad_in.data_ptr<scalar_t>(),
          rois.data_ptr<scalar_t>());
  }));

  AT_ASSERT(hipGetLastError() == hipSuccess);
  return grad_in;
}
