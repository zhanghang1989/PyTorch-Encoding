#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include "ATen/NativeFunctions.h"
#include <ATen/cuda/HIPContext.h>

template <typename scalar>
__device__ __forceinline__ scalar fmin(scalar a, scalar b) {
  return a > b ? b : a;
}

template <typename scalar>
__device__ __forceinline__ scalar fmax(scalar a, scalar b) {
  return a > b ? a : b;
}

template <typename scalar>
__device__ __forceinline__ scalar IoU(const scalar* box_x, const scalar* box_y) {
  // Calculate IoU between the boxes.
  scalar rightmost_l = fmax(box_x[0], box_y[0]);
  scalar leftmost_r = fmin(box_x[0] + box_x[2], box_y[0] + box_y[2]);
  scalar delta_x = fmax((scalar)0., leftmost_r - rightmost_l);

  scalar bottommost_tp = fmax(box_x[1], box_y[1]);
  scalar topmost_b = fmin(box_x[1] + box_x[3], box_y[1] + box_y[3]);
  scalar delta_y = fmax((scalar)0., topmost_b - bottommost_tp);

  scalar uni = box_x[2] * box_x[3] + box_y[2] * box_y[3];

  return delta_x * delta_y / (uni - delta_x * delta_y);

}

template <typename scalar>
__global__ void nms_kernel(unsigned char* mask, 
                          const scalar* boxes,
                          const int64_t* inds,
                          const int64_t num_boxes,
                          double thresh) {
//A pretty straightforward implementation, analogous to the standard serial
//version but with the IoUs computed and mask updated in parallel. We access
//the box data through an array of sorted indices rather than physically
//sorting it: unless one has an inordinate number of boxes (O(10^5), whereas
//for example in the faster rcnn paper they feed 6000 per batch) the
//data will fit in L2 so sorting it won't actually reduce the number of
//messy reads from global.
  int col = 0;
  while(col < num_boxes-1)
  {
    for(int i = threadIdx.x; i < num_boxes-1; i+=blockDim.x)
      if(i >= col)
      {
        scalar iou = IoU(&boxes[4*inds[i+1+num_boxes*blockIdx.x] + 4*num_boxes*blockIdx.x],
                         &boxes[4*inds[col+num_boxes*blockIdx.x] + 4*num_boxes*blockIdx.x]);
        mask[i+1+blockIdx.x*num_boxes] *= (iou>thresh) ? 0 : 1;
      }
    __syncthreads();
    ++col;
    while((col < num_boxes - 1) && (mask[col+blockIdx.x*num_boxes]==0))
      ++col;
  }
}

std::vector<at::Tensor> Non_Max_Suppression_CUDA(
    const at::Tensor& input,
    const at::Tensor& scores,
    double thresh) {
  AT_ASSERT(input.ndimension() == 3);
  AT_ASSERT(scores.ndimension() == 2);
  AT_ASSERT(input.size(0) == scores.size(0));
  AT_ASSERT(input.size(1) == scores.size(1));
  AT_ASSERT(input.size(2) == 4);
  AT_ASSERT(input.is_contiguous());
  AT_ASSERT(scores.is_contiguous());
  AT_ASSERT(input.scalar_type() == at::kFloat || input.scalar_type() == at::kDouble);
  AT_ASSERT(scores.scalar_type() == at::kFloat || scores.scalar_type() == at::kDouble);

  auto num_boxes = input.size(1);
  auto batch_size = input.size(0);
  //auto mask = input.type().toScalarType(at::kByte).tensor({batch_size, num_boxes});
  auto mask = torch::zeros({batch_size, num_boxes}, input.type().toScalarType(at::kByte));
  mask.fill_(1);
  
  //need the indices of the boxes sorted by score.
  at::Tensor sorted_inds = std::get<1>(scores.sort(-1, true));


  dim3 mask_block(512); //would be nice to have 1024 here for gpus that support it,
                        //but not sure how to do this cleanly without calling
                        //hipGetDeviceProperties in the funcion body...

  dim3 mask_grid(batch_size);
  if(input.scalar_type() == at::kFloat)
  {
      nms_kernel<<<mask_grid, mask_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                                        mask.data_ptr<unsigned char>(),
                                        input.data_ptr<float>(),
                                        sorted_inds.data_ptr<int64_t>(),
                                        num_boxes,
                                        thresh);
      AT_ASSERT(hipGetLastError() == hipSuccess);
  }
  else
  {
      nms_kernel<<<mask_grid, mask_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                                        mask.data_ptr<unsigned char>(),
                                        input.data_ptr<double>(),
                                        sorted_inds.data_ptr<int64_t>(),
                                        num_boxes,
                                        thresh);
      AT_ASSERT(hipGetLastError() == hipSuccess);
  }

  //It's not entirely clear what the best thing to return is here. The algorithm will
  //produce a different number of boxes for each batch, so there is no obvious way of
  //way of returning the surving boxes/indices as a tensor. Returning a mask on the
  //sorted boxes together with the sorted indices seems reasonable; that way, the user
  //can easily take the N highest-scoring surviving boxes to form a tensor if they wish. 
  return {mask, sorted_inds};
}
